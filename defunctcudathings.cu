#include "hip/hip_runtime.h"

__global__ void dotProduct(int *d_baseDenseRIV,
 							int *d_multiplierBlock, int *d_multiplierValueCount, 
							int *d_output, int multiplierCount, int displacement){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id>=multiplierCount) return;
	int *d_multiplierLocations = d_multiplierBlock +(id*2*displacement);
	int *d_multiplierValues = d_multiplierLocations+displacement;
	//int *d_baseStop = d_baseLocations+baseValueCount;
	int *d_multiplierStop = d_multiplierLocations+(d_multiplierValueCount[id]);
	//printf("spacing: %d, %d\n", (d_multiplierStop-d_multiplierLocations), d_multiplierValueCount[id]);
	d_output+=id;
	*d_output= 0;	
	while(d_multiplierLocations< d_multiplierStop){
		*d_output += (*d_multiplierValues)*(d_baseDenseRIV[*d_multiplierLocations]);
		d_multiplierValues++;
		d_multiplierLocations++;
		
	}
}
__global__ void getMagnitude(float *d_magnitudes, int *d_values, int *valueCount, int RIVCount, int memSectionSize){
	//consider changing to single operation per thread
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id>=RIVCount) return;
	d_values+=(id*(memSectionSize));
	int *stop = d_values+ valueCount[id];
	float *magnitude = d_magnitudes+id;
	*magnitude = 0;
	for( ;d_values<stop; d_values++){
		*magnitude +=(*d_values)*(*d_values);
	}
	
	*magnitude = sqrt(*magnitude);
	
}
int* getDotProducts(sparseRIV *inputs, int baseNumber, int RIVCount, int maxSize){
	
	int remainingSet = RIVCount-(baseNumber+1);
	int *output = (int*)malloc(remainingSet*sizeof(int));
	int *output_slider=output;
	//badly written, fix it
	int *d_baseDenseRIV = RIVKeyData.d_OpenSlot;
	int *baseDenseRIV = mapS2D(inputs[baseNumber]);//as a byproduct, also places a denseRIV form of the input into RIVKeyData.d_OpenSlot at the beginning;
	
	HANDLE_ERROR (hipMemcpy (d_baseDenseRIV, baseDenseRIV, RIVKeyData.RIVsize*sizeof(int), hipMemcpyHostToDevice));
	int *d_slider = RIVKeyData.d_OpenSlot+RIVKeyData.RIVsize;

	
	int *valueCounts = (int*)malloc(remainingSet*sizeof(int));
	
	int *d_valueCounts = d_slider;
	d_slider+=remainingSet;
	int *d_output = d_slider;
	d_slider+=remainingSet;
	int *d_multiplierBlock = d_slider;
	
	int i=baseNumber+1;
	while(i<RIVCount){
		int doneSoFar = i;
		while((i<RIVCount) && (d_slider < RIVKeyData.d_SlotEnd)){
			if(inputs[i].boolean){
				//each set of locations and then values is layed out linear in GPU ram with buffer the size of the largest RIV
				HANDLE_ERROR (hipMemcpy (d_slider, inputs[i].locations, inputs[i].count*sizeof(int), hipMemcpyHostToDevice));
				d_slider +=maxSize;
				HANDLE_ERROR (hipMemcpy (d_slider, inputs[i].values, inputs[i].count*sizeof(int), hipMemcpyHostToDevice));
				d_slider +=maxSize;
				//printf("%p", d_slider);
				valueCounts[i-doneSoFar] = inputs[i].count;
			}else{
				valueCounts[i-doneSoFar] = 0;
			}
			i++;
		}
		int thisBlock = i-doneSoFar;
		HANDLE_ERROR (hipMemcpy (d_valueCounts, valueCounts, thisBlock*sizeof(int), hipMemcpyHostToDevice));
		//d_slider+= remainingSet;

		int blockSize;  
		int minGridSize = 0;
		int gridSize; 
		hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, dotProduct); 
		gridSize = ((thisBlock + blockSize -1) / blockSize)+1; 

			dotProduct<<<gridSize,blockSize>>>(d_baseDenseRIV, d_multiplierBlock, d_valueCounts, 
																d_output, thisBlock, maxSize);
		
		
		HANDLE_ERROR (hipMemcpy (output_slider, d_output, thisBlock*sizeof(int), hipMemcpyDeviceToHost));
		//printf("did a thing");
		output_slider+=thisBlock;
		d_slider = d_multiplierBlock;
		
	
	}
	return output;
	
}
float* getMagnitudes(sparseRIV *dataSet, int RIVCount, int maxSize){
	//int **values = (int**)malloc(RIVCount*sizeof(int*));
	/*int **d_values;
	HANDLE_ERROR (hipMalloc((void***)&d_values, RIVCount*sizeof(int*)));
	*/
	int *valueCounts = (int*)malloc(RIVCount*sizeof(int));
	float *magnitudes = (float*)malloc(RIVCount*sizeof(float));
	HANDLE_ERROR (hipMalloc((void**)&RIVKeyData.d_magnitudes, RIVCount*sizeof(float)));
	//HANDLE_ERROR(hipMemset(RIVKeyData.d_magnitudes, 0, RIVCount*(sizeof(float))));
	float *magnitudes_slider = magnitudes;
	
	int *d_slider = RIVKeyData.d_OpenSlot;//+RIVCount;
	
	int *d_valueCounts = d_slider;
	d_slider+=RIVCount;
	int *d_valuesBlock = d_slider;
	//printf("magnitudesSlot: %d, %d\n", d_slider-RIVKeyData.d_OpenSlot, RIVCount*sizeof(float));
	//prepare for overflow?
	//printf("%d\n", RIVCount);
	int i=0;
	while(i<RIVCount){
		int doneSoFar = i;
		
		while((d_slider<RIVKeyData.d_SlotEnd) && (i<RIVCount)){
			HANDLE_ERROR (hipMemcpy (d_slider, dataSet[i].values, dataSet[i].count*sizeof(int), hipMemcpyHostToDevice));
			d_slider +=maxSize;
			valueCounts[i-doneSoFar] = dataSet[i].count;
			i++;
			//printf("%d, %d, %d, %d\n", d_slider-RIVKeyData.d_OpenSlot, doneSoFar, i, RIVCount);
		}
		int thisBlock = i-doneSoFar;
		
		//HANDLE_ERROR (hipMemcpy (d_values, values, RIVCount*sizeof(int*), hipMemcpyHostToDevice));
		HANDLE_ERROR (hipMemcpy (d_valueCounts, valueCounts, thisBlock*sizeof(int), hipMemcpyHostToDevice));
		int blockSize;  
		int minGridSize = 0;
		int gridSize; 
		hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, getMagnitude); 

		
		//HANDLE_ERROR (hipMalloc((void**)&d_magnitudes, RIVCount*sizeof(float)));
		
		
		gridSize = ((thisBlock + blockSize -1) / blockSize)+1; 
			
			
			getMagnitude<<<gridSize,blockSize>>>(RIVKeyData.d_magnitudes, d_valuesBlock, d_valueCounts, thisBlock, maxSize);
		//printf("got here");
		HANDLE_ERROR (hipMemcpy (magnitudes_slider, RIVKeyData.d_magnitudes, thisBlock*sizeof(float), hipMemcpyDeviceToHost));
		magnitudes_slider+=thisBlock;
		d_slider =d_valuesBlock;
	}
	for(int i=0; i<RIVCount; i++){
		dataSet[i].magnitude = magnitudes[i];
		//printf("%f\n", dataSet[i].magnitude);
	}
	return magnitudes;
}
sparseRIV compileD2SOrdered(denseRIV input){
	
	//int *valueCount;
	//*RIVsize = 0;
	int *d_valueCount;
	HANDLE_ERROR(hipMalloc((void**)&d_valueCount, sizeof(int)));
	HANDLE_ERROR(hipMemset(d_valueCount, 0, sizeof(int)));
	int *d_locations = RIVKeyData.d_OpenSlot+RIVKeyData.RIVsize;
	//HANDLE_ERROR (hipMemcpy (d_valueCount, valueCount, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR (hipMemcpy (RIVKeyData.d_OpenSlot, input.values, RIVKeyData.RIVsize*sizeof(int), hipMemcpyHostToDevice));
	int blockSize;  
	int minGridSize = 0;
	int gridSize; 
	hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, D2SLocations); 
	
	gridSize = ((RIVKeyData.RIVsize + blockSize -1) / blockSize)+1; 
		D2SLocations <<<gridSize,blockSize>>> (RIVKeyData.d_OpenSlot, d_locations, d_valueCount, RIVKeyData.RIVsize);
		hipDeviceSynchronize();
	sparseRIV output;
	HANDLE_ERROR (hipMemcpy (&output.count, d_valueCount, sizeof(int), hipMemcpyDeviceToHost));

	output.values = (int*)malloc(output.count*sizeof(int));
	if(output.values ==NULL) printf("malloc fail 246");
	output.locations = (int*)malloc(output.count*sizeof(int));
	if(output.locations ==NULL) printf("malloc fail 248");
	HANDLE_ERROR (hipMemcpy (output.locations, d_locations, (output.count)*sizeof(int), hipMemcpyDeviceToHost));
	qsort(output.locations, output.count, sizeof(int), compareLocations);
	
	
	for(int i=0; i<output.count; i++){
		output.values[i] = input.values[output.locations[i]];
	}
	free(input.values);
	hipFree(d_valueCount);
	return output;
	
}
int compareLocations(const void *first, const void *second){
	int *f = (int*)first;
	int *s = (int*)second;
	return(*f - *s);
}
__global__ void D2SLocations(int *d_DenseRIV, int* d_SparseLocations, int* d_NZCount, int d_DenseSize){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>=d_DenseSize) return;
	if(!d_DenseRIV[id]) return;
	int sparseSlot = atomicAdd(d_NZCount, 1);
	d_SparseLocations[sparseSlot] = id;
}	
