#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "RIVCull.h"
int main(int argc, char argv[]){
	clock_t beginprep = clock();
	HANDLE hfind = NULL;
	WIN32_FIND_DATA findFile;
	int fileCount = 0;
	setKeyData(25000, 4);
	sparseRIV *fileRIVs = (sparseRIV*) malloc(7000*sizeof(sparseRIV));	
	//switch to dynamic
	char pathString[2000];
	sprintf(pathString, "%s\\*.*","0");
	if((hfind = FindFirstFile(pathString, &findFile)) == INVALID_HANDLE_VALUE){
		printf("invalid path: %s\n", pathString);
		return 1;
	}
	int maxSize = 0;
	do{
		if(strcmp(findFile.cFileName, ".") != 0 && 
		strcmp(findFile.cFileName, "..") != 0){
			sprintf(pathString, "%s\\%s", "0", findFile.cFileName);
			FILE *input = fopen(pathString, "r");
			//printf("%s\n", pathString);
			fileRIVs[fileCount] = FileToL2(input);
			strcpy(fileRIVs[fileCount].name, pathString);
			if(fileRIVs[fileCount].count>maxSize){
				maxSize = fileRIVs[fileCount].count;
			}
			/*for(int i=0; i<fileRIVs[fileCount].count; i++){
				printf("%d, %d\n", fileRIVs[fileCount].locations[i], fileRIVs[fileCount].values[i]);
			}*/
			fclose(input);
			fileCount++;
		}
	}while(FindNextFile(hfind, &findFile));
	

	FindClose(hfind);
	float *magnitudes = getMagnitudesCPU(fileRIVs, fileCount); 

	clock_t beginnsquared = clock();

	float **cosSims = (float**) malloc(fileCount*sizeof(float*));
	for(int i=0; i<fileCount; i++){
		if(fileRIVs[i].boolean){
			*cosSims = cosineCompare(fileRIVs[i], fileRIVs[i+1], fileCount-(i+1));
		}
		cosSims++;
		
	}
	clock_t endnsquared = clock();
	double time = (double)(endnsquared - beginnsquared) / CLOCKS_PER_SEC;
	printf("nsquared time:%lf\n\n", time);
	printf("%d <", RIVKeyData.thing);
		clock_t endprep = clock();
	double time_spent = (double)(endprep - beginprep) / CLOCKS_PER_SEC;
	printf("total time:%lf\n\n", time_spent);
return 0;
}
